#include "hip/hip_runtime.h"
#include "labeling.h"

const int threads = 256;

__device__ __host__ int CeilDiv(int a, int b) { return (a - 1) / b + 1; }

__global__ void add_aux(int *pos, int text_size, int *d_aux) {
  int gindex = blockIdx.x * blockDim.x + threadIdx.x;
  int lindex = threadIdx.x;

  if (blockIdx.x != 0 && gindex < text_size)
    if (lindex == pos[gindex] - 1)
      pos[gindex] += d_aux[blockIdx.x - 1];
}

__global__ void intra_block_scan(const char *text, int *pos, int text_size, int *d_aux) {
  __shared__ int sdata[threads * 2];
  __shared__ bool fdata[threads * 2];
  __shared__ bool ofdata[threads * 2];
  __shared__ bool add1[threads * 2];

  int gindex = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
  int lindex = threadIdx.x * 2;
  int tid = threadIdx.x;

  if (gindex < text_size)
    sdata[lindex] = add1[lindex] = (text[gindex] == ' ' ? 0 : 1);
  if (gindex + 1 < text_size)
    sdata[lindex + 1] = add1[lindex + 1] = (text[gindex + 1] == ' ' ? 0 : 1);
  __syncthreads();

  if (lindex == 0)
    fdata[lindex] = ofdata[lindex] = true;
  else
    fdata[lindex] = ofdata[lindex] = (sdata[lindex - 1] != sdata[lindex]);
  fdata[lindex + 1] = ofdata[lindex + 1] = (sdata[lindex] != sdata[lindex + 1]);

  int offset = 1;
  for (int d = threads; d > 0; d >>= 1) {
    __syncthreads();
    if (tid < d) {
      int ai = offset * (lindex + 1) - 1;
      int bi = offset * (lindex + 2) - 1;
      if (!fdata[bi])
        sdata[bi] += sdata[ai];
      fdata[bi] = fdata[ai] || fdata[bi];
    }
    offset <<= 1;
  }

  if (tid == 0)
    sdata[threads * 2 - 1] = 0;

  for (int d = 1; d <= threads; d <<= 1) {
    __syncthreads();
    offset >>= 1;
    if (tid < d) {
      int ai = offset * (lindex + 1) - 1;
      int bi = offset * (lindex + 2) - 1;
      int t = sdata[ai];
      sdata[ai] = sdata[bi];
      if (ofdata[ai + 1])
        sdata[bi] = 0;
      else if (fdata[ai])
        sdata[bi] = t;
      else
        sdata[bi] += t;
      fdata[ai] = false;
    }
  }
  __syncthreads();

  if (gindex < text_size)
    pos[gindex] = sdata[lindex] + add1[lindex];
  if (gindex + 1 < text_size)
    pos[gindex + 1] = sdata[lindex + 1] + add1[lindex + 1];

  if (tid == 0)
    d_aux[blockIdx.x] = sdata[threads * 2 - 1] + add1[threads * 2 - 1];
}

void labeling(const char *text, int *pos, int text_size) {
  int *d_aux;
  hipMalloc(&d_aux, CeilDiv(text_size, threads) * sizeof(int));
  intra_block_scan<<<CeilDiv(text_size, threads * 2), threads>>>(
      text, pos, text_size, d_aux);
  add_aux<<<CeilDiv(text_size, threads * 2), threads * 2>>>(pos, text_size,
                                                            d_aux);
  hipFree(d_aux);
}
